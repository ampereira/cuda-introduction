/**************************************************************
 * 
 * --== Simple CUDA kernel ==--
 * author: ampereira
 * 
 *
 * Fill the rest of the code through the following steps:
 * -> allocate the device memory
 * -> copy the inputs to the device
 * -> call the kernel
 * -> copy the output to the host
 * 
 * extra points for
 * -> reversing the output array on the device
 *
 **************************************************************/


#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

#define NUM_BLOCKS 16
#define NUM_THREADS_PER_BLOCK 32
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK

using namespace std;

void checkCUDAError (const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
		exit(-1);
	}
}

// Fill the input parameters and kernel qualifier
void vecAdditionKernel () {

}

int main( int argc, char** argv) {
	// arrays on the host
	float a[SIZE], b[SIZE], c[SIZE];

	// pointers to the device memory


	// fills the arrays
	for (unsigned i = 0; i < SIZE; ++i) {
		a[i] = rand() / RAND_MAX;
		b[i] = rand() / RAND_MAX;
	}

	// allocate the memory on the device

	checkCUDAError("mem allocation");
	// copy inputs to the device

	checkCUDAError("memcpy h->d");
	// launch the kernel

	checkCUDAError("kernel invocation");
	// copy the output to the host

	checkCUDAError("memcpy d->h");
	// free the device memory

	checkCUDAError("mem free");

	return 0;
}